// ----------------------------------------------------------------
// Sudoku -- Puzzle Solver on GPU using CUDA
// ----------------------------------------------------------------

/**
 * @file
 * sudoku.cu
 *
 * @brief main sudoku file to init and execute
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// includes, kernels
// #include "kernels.cuh"

// includes, utilities
#include "util/error_utils.cuh"
#include "util/io_utils.cuh"
#include "data.cuh"


int main(int argc, char** argv) {

    /* Gets arguments from command line and puzzle from a file */
    CommandLineArgs * build = new CommandLineArgs;
    input(argc, argv, build);
    /* KernelManager((*build).size, &(*build).unsolved, (*build).graphpics); */

}

void KernelManager(int n, Square * h_unsolved, bool o_graphics) {

  /* CUDA event setup */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Memory Allocations */
  int memsize = sizeof(Square) * n * n;

  Square * d_unsolved;
  ERROR_CHECK( hipMalloc((void**) &d_unsolved, memsize) );
  ERROR_CHECK( hipMemcpy(d_unsolved, h_unsolved, memsize,
                          hipMemcpyHostToDevice) );

  Square * d_solved;
  ERROR_CHECK( hipMalloc((void**) &d_solved, memsize) );

  float elapsedTime;
  hipEventRecord(start, 0);

  // TODO: Kernel execution
  // TODO: All of them can go one by one,
  // TODO: we'll just need to reset event record,
  // TODO: for multiple timing/performance measurements.

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  Square * h_solved = (Square *) malloc(memsize);
  ERROR_CHECK( hipMemcpy(h_solved, d_solved, memsize,
                          hipMemcpyDeviceToHost) );

  /* Destroy CUDA event */
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // TODO: Terminal Output will go here.

  /* Free Memory Allocations */
  free(h_unsolved);
  ERROR_CHECK( hipFree(d_unsolved) );
  ERROR_CHECK( hipFree(d_solved) );
}
