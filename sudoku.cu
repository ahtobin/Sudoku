// ----------------------------------------------------------------
// Sudoku -- Puzzle Solver on GPU using CUDA
// ----------------------------------------------------------------

/**
 * @file
 * sudoku.cu
 *
 * @brief main sudoku file to init and execute
 */

#pragma once

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// includes, kernels
#include "kernels.cuh"

// includes, utilities
#include "util/error_utils.cuh"
#include "util/io_utils.cuh"


int main(int argc, char** argv) {

    input(argc, argv);

}
