// ----------------------------------------------------------------
// Sudoku -- Puzzle Solver on GPU using CUDA
// ----------------------------------------------------------------

/**
 * @file
 * sudoku.cu
 *
 * @brief main sudoku file to init and execute
 */

#pragma once

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// includes, kernels
#include "kernels.cuh"

// includes, utilities
#include "util/error_utils.cuh"
#include "util/io_utils.cuh"
#include "data.cuh"


int main(int argc, char** argv) {

    /* Gets arguments from command line and puzzle from a file */
    CommandLineArgs * build = new CommandLineArgs;
    input(argc, argv, build);
    KernelManager((*build).size, &(*build).unsolved, (*build).graphpics);

}

void KernelManager(Size n, Puzzle * h_unsolved, bool o_graphics) {

  /* CUDA event setup */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Memory Allocations */
  memsize = sizeof(Puzzle) * (*build).size * (*build).size;

  Puzzle * d_unsolved;
  ERROR_CHECK( hipMalloc((void**) &d_unsolved, memsize) );
  ERROR_CHECK( hipMemcpy(d_unsolved, h_unsolved, memsize,
                          hipMemcpyHostToDevice) );

  float elapsedTime;
  hipEventRecord(start, 0);

  // TODO: Kernel execution
  // TODO: All of them can go one by one,
  // TODO: we'll just need to reset event record,
  // TODO: for multiple timing/performance measurements.

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  Puzzle * d_solved;
  ERROR_CHECK( hipMalloc((void**) &d_solved, memsize) );

  /* Destroy CUDA event */
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // TODO: Terminal Output will go here.

  /* Free Memory Allocations */
  free(h_unsolved);
  ERROR_CHECK( hipFree(d_unsolved) );
  ERROR_CHECK( hipFree(d_solved) );
}
